// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Paralel Systems
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage
// Multiplication of elements in float array
//
// ***********************************************************************

#include <hip/hip_runtime.h>
#include <stdio.h>

// Demo kernel for array elements multiplication.
// Every thread selects one element and multiply it. 
__global__ void kernel_mult( float *pole1, float *pole2, int L)
{
	int l = blockDim.x * blockIdx.x + threadIdx.x;
	// if grid is greater then length of array...
	if (l>=L) return;
	for(int i=0;i<30;i++)
	{
	pole1[l]*=pole2[l];
	pole1[l]/=pole2[l];
	}
	pole1[l] *= pole2[l];
}

void run_mult( float *P1, float *P2, int Length)
{
	hipError_t cerr;
	int threads = 1024;
	int blocks = ( Length + threads - 1 ) / threads;
	printf("blocks: %d\n", blocks);
	// Memory allocation in GPU device
	float *cudaP1;
	float *cudaP2;
	cerr = hipMalloc( &cudaP1, Length * sizeof( float ) );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	
	
	cerr = hipMalloc( &cudaP2, Length * sizeof( float ) );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	

	// Copy data from PC to GPU device
	cerr = hipMemcpy( cudaP1, P1, Length * sizeof( float ), hipMemcpyHostToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	
	cerr = hipMemcpy( cudaP2, P2, Length * sizeof( float ), hipMemcpyHostToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	

	// Grid creation
	kernel_mult<<< blocks, threads >>>(cudaP1, cudaP2, Length);

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Copy data from GPU device to PC
	cerr = hipMemcpy( P1, cudaP1, Length * sizeof( float ), hipMemcpyDeviceToHost );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Free memory
	hipFree(cudaP1);
	hipFree(cudaP2);
}

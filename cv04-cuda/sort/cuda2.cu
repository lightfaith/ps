// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Paralel Systems
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage
// Multiplication of elements in float array
//
// ***********************************************************************

#include <hip/hip_runtime.h>
#include <stdio.h>

//#define swap(float *p1, float *p2){	float tmp = *p1;	*p1=*p2;	*p2=tmp;}

// Demo kernel for array elements multiplication.
// Every thread selects one element and multiply it. 
__global__ void kernel_mult( float *pole, int L, int inc)
{
	// No 2 swapping in one kernel (collisions), rather run 2 kernels in loop below...
	int l = blockDim.x * blockIdx.x + threadIdx.x;
	if(l%2==1)
		return;
	// if grid is greater then length of array...
	int border = (L-1-inc);

	if (l>=border) return;
	float tmp;
	if(pole[l+inc]>pole[l+1+inc])
	{
		tmp=pole[l+inc];
		pole[l+inc]=pole[l+1+inc];
		pole[l+1+inc]=tmp;
	}
}

void bsort( float *P, int Length)
{
	hipError_t cerr;
	int threads = 1024;
	int blocks = ( Length + threads - 1 ) / threads;
	printf("blocks: %d\n", blocks);
	// Memory allocation in GPU device
	float *cudaP;
	cerr = hipMalloc( &cudaP, Length * sizeof( float ) );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	
	

	// Copy data from PC to GPU device
	cerr = hipMemcpy( cudaP, P, Length * sizeof( float ), hipMemcpyHostToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	

	// Grid creation
	for(int i=0;i<Length/2; i++)
	{
		kernel_mult<<< blocks, threads >>>(cudaP, Length, 0);
		kernel_mult<<< blocks, threads >>>(cudaP, Length, 1);
	}

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Copy data from GPU device to PC
	cerr = hipMemcpy( P, cudaP, Length * sizeof( float ), hipMemcpyDeviceToHost );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Free memory
	hipFree(cudaP);
}

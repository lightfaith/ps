// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Paralel Systems
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage
// Multiplication of elements in float array
//
// ***********************************************************************

#include <hip/hip_runtime.h>
#include <stdio.h>

//#define swap(float *p1, float *p2){	float tmp = *p1;	*p1=*p2;	*p2=tmp;}

// Demo kernel for array elements multiplication.
// Every thread selects one element and multiply it. 
__global__ void kernel_mult( float *pole, int L)
{
	// No 2 swapping in one kernel (collisions), rather run 2 kernels in loop below...
	int l = blockDim.x * blockIdx.x + threadIdx.x;
	if(l%2==1)
		return;
	// if grid is greater then length of array...
	if (l>=L-1) return;
	float tmp;
	if(pole[l]>pole[l+1])
	{
		tmp=pole[l];
		pole[l]=pole[l+1];
		pole[l+1]=tmp;
	}
	if(l>=L-2) return;
	if(pole[l+1]>pole[l+2])
	{
		tmp=pole[l+1];
		pole[l+1]=pole[l+2];
		pole[l+2]=tmp;
	}
}

void bsort( float *P, int Length)
{
	hipError_t cerr;
	int threads = 1024;
	int blocks = ( Length + threads - 1 ) / threads;
	printf("blocks: %d\n", blocks);
	// Memory allocation in GPU device
	float *cudaP;
	cerr = hipMalloc( &cudaP, Length * sizeof( float ) );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	
	

	// Copy data from PC to GPU device
	cerr = hipMemcpy( cudaP, P, Length * sizeof( float ), hipMemcpyHostToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	

	// Grid creation
	for(int i=0;i<Length; i++)
	// here run kernel twice
		kernel_mult<<< blocks, threads >>>(cudaP, Length);

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Copy data from GPU device to PC
	cerr = hipMemcpy( P, cudaP, Length * sizeof( float ), hipMemcpyDeviceToHost );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Free memory
	hipFree(cudaP);
}
